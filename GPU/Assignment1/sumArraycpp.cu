#include "hip/hip_runtime.h"
// C++ version of sumarray from GPU course slides
// Compile: nvcc -I "C:\ProgramData\NVIDIA Corporation\CUDA Samples\v10.1\common\inc" sumArraycpp.cu -o sumarraycpp
// Run: sumarraycpp

#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <hip/hip_runtime_api.h>

using namespace std;

/*
 * This example implements Array addition on the host and GPU.
 * sumArrayOnHost iterates over the elements, adding
 * elements from A and B together and storing the results in C. 
 * sumArrayOnGPU implements the same logic, but using CUDA threads to process each element.
 */


void sumArrayOnHost(float *A, float *B, float *C, const int n)
{
    float *ia = A;
    float *ib = B;
    float *ic = C;

     for (int ix = 0; ix < n; ix++)
        {
            ic[ix] = ia[ix] + ib[ix];
        }
    return;
}


__global__ void sumArrayOnGPU(float *A, float *B, float *C, int N)
{
    unsigned int idx = threadIdx.x;
 
    if (idx < N)
    {
        C[idx] = A[idx] + B[idx];
    }
}

#define nelem 5000

void checkerror(hipError_t err)
{ if (err != hipSuccess) cout << hipGetErrorString(err) << endl;
	return; 
}


int main(int argc, char **argv)
{
    int nBytes = nelem * sizeof(float);

    // malloc host memory
    float *h_A, *h_B, *hostC, *gpuC;
    h_A = new float[nelem];
    h_B = new float[nelem];
    hostC = new float[nelem];
    gpuC = new float[nelem];

    // initialise A and B
     for (int i=0; i < nelem; i++)
      { h_A[i] = i;
        h_B[i] = i; }

    memset(hostC, 0, nBytes);
    memset(gpuC, 0, nBytes);

    // add Array at host side for result checks
    sumArrayOnHost (h_A, h_B, hostC, nelem);
   cout << "Host sum is: " << hostC[9] << endl;

	// malloc device global memory
    float *d_A , *d_B, *d_C;
    checkerror(hipMalloc((void **)&d_A , nBytes)); 
    checkerror(hipMalloc((void **)&d_B, nBytes)); 
    checkerror(hipMalloc((void **)&d_C, nBytes)); 

     // transfer data from host to device
    checkerror(hipMemcpy(d_A , h_A, nBytes, hipMemcpyHostToDevice));
    checkerror(hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice));

	// execute the kernel
    checkerror(hipDeviceSynchronize());
    sumArrayOnGPU<<<10,nelem/10>>>(d_A, d_B, d_C, nelem);
    checkerror(hipGetLastError());

 
    // copy kernel result back to host side
    checkerror(hipMemcpy(gpuC, d_C, nBytes, hipMemcpyDeviceToHost));
	cout << "GPU sum is: " << gpuC[9] << endl;
	
     // free device global memory
    checkerror(hipFree(d_A ));
    checkerror(hipFree(d_B));
    checkerror(hipFree(d_C));

    // free host memory
    free(h_A);
    free(h_B);
    free(hostC);
    free(gpuC);

    // reset device
    checkerror(hipDeviceReset());

    return 0;
}
